/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <stdexcept>

namespace vt { namespace cuda {

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        throw std::logic_error("cuda API failed");
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        throw std::logic_error("cuBLAS API failed");
    }
}


/// Sample wrapper executing single precision gemm with cublasLtMatmul, nearly a drop-in replacement for cublasSgemm,
/// with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
/// this configure appropriate attribute in the preference handle
void lt_sgemm(hipblasLtHandle_t ltHandle,
             hipblasOperation_t transa,
             hipblasOperation_t transb,
             int m,
             int n,
             int k,
             const float *alpha, /* host pointer */
             const void *A, hipDataType at,
             int lda,
             const void *B, hipDataType bt, 
             int ldb,
             const float *beta, /* host pointer */
             void *C, hipDataType ct,
             int ldc,
             void *workspace,
             size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see cublasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, at, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, bt, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, ct, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from cudaMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     beta,
                                     C,
                                     Cdesc,
                                     C,
                                     Cdesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}

/// Sample wrapper executing single precision gemm with cublasLtMatmul, nearly a drop-in replacement for cublasSgemm,
/// with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
/// this configure appropriate attribute in the preference handle
void lt_sgemm_batched(hipblasLtHandle_t ltHandle,
             hipblasOperation_t transa,
             hipblasOperation_t transb,
             int m,
             int n,
             int k,
             const float *alpha, /* host pointer */
             const void *A, hipDataType at,
             int lda,
             const void *B, hipDataType bt, 
             int ldb,
             const float *beta, /* host pointer */
             void *C, hipDataType ct,
             int ldc,
             int batchCount,
             void *workspace,
             size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    int64_t stridea = m*k;
    int64_t strideb = n*k;
    int64_t stridec = m*n;

    // create operation desciriptor; see cublasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, at, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, bt, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, ct, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec)));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from cudaMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }
   
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     beta,
                                     C,
                                     Cdesc,
                                     C,
                                     Cdesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));


    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}



}} // endof cuda
